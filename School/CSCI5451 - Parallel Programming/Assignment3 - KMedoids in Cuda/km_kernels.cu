#include "hip/hip_runtime.h"
﻿
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "Point.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <stdio.h>


__device__ double distanceTo(const double* p1, const double* p2, int dimensions) {
    double sum = 0.0;
    for (int i = 0; i < dimensions; i++) {
        double distComp = p1[i] - p2[i];
        sum += (distComp * distComp);
    }
    return sum;
}

__global__ void setCoords(int clusterId, hipcub::KeyValuePair<int, double>* kvp, Point* medoids, Point* allPoints, thrust::device_vector<int> pointIndices) {
    int minIndex = kvp->key;
    int medoidInd = pointIndices[minIndex];
    medoids[clusterId].coordinates = allPoints[medoidInd].coordinates;
}

__global__ void clusterPoints(int numMedoids, int numPoints, Point* medoids, Point* allPoints, int dimensions, int* &medoidInds, bool &convergence) {
    extern __shared__ Point sharedPoints[];
    for (int p = threadIdx.x; p < numMedoids; p += blockDim.x) {
        sharedPoints[p] = medoids[p];
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < numPoints; i += stride) {
        double minDistance = 1000000000.0;
        int minDistInd = 0;
        for (int j = 0; j < numMedoids; j++) {
            double distance = distanceTo(allPoints[i].coordinates, sharedPoints[j].coordinates, dimensions);
            if (distance <= minDistance) {
                minDistance = distance;
                minDistInd = j;
            }
        }
        if (medoidInds[i] != minDistInd) {
            medoidInds[i] = minDistInd;
            convergence = false;
        }
    }
}

__global__ void flagPointsInCluster(int numPoints, int clusterId, int* medoidInds, thrust::device_vector<int> &pointFlags) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < numPoints; i += stride) {
        if (medoidInds[i] == clusterId) {
            pointFlags[i] = 1;
        }
    }
}

__global__ void generateDists(int dimensions, int clusterId, Point* allPoints, double* &dists, int pointsInCluster, thrust::device_vector<int> pointIndices) {
    extern __shared__ Point sharedPoints[];
    for (int p = threadIdx.x; p < pointsInCluster; p += blockDim.x) {
        int pointIndex = pointIndices[p];
        sharedPoints[p] = allPoints[pointIndex];
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int p = index; p < pointsInCluster; p += stride) {
        double totalDist = 0.0;
        for (int p2 = 0; p2 < pointsInCluster; p2++) {
            totalDist += distanceTo(sharedPoints[p].coordinates, sharedPoints[p2].coordinates, dimensions);
        }
        dists[p] = totalDist;
    }
}

void callClusterPointsKernel(int numMedoids, int numPoints, Point* medoids, Point* allPoints, int dimensions, int* &medoidInds, bool &convergence, int numBlocks, int threadsPerBlock) {
    size_t sharedMemorySize = numMedoids * sizeof(Point);
    clusterPoints<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(numMedoids, numPoints, medoids, allPoints, dimensions, medoidInds, convergence);
}

void callFlagPointsInClusterKernel( int numPoints, int clusterId, int* medoidInds, thrust::device_vector<int> &pointFlags, int numBlocks, int threadsPerBlock) {
    flagPointsInCluster<<<numBlocks, threadsPerBlock>>>(numPoints, clusterId, medoidInds, pointFlags);
}

void callGenerateDistsAndAssignMedoidsKernel(int dimensions, int clusterId, Point* allPoints, Point* medoids, double* &dists, int pointsInCluster, thrust::device_vector<int> pointIndices, int numBlocks, int threadsPerBlock) {
    size_t sharedMemorySize = pointsInCluster * sizeof(Point);
    generateDists<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(dimensions, clusterId, allPoints, dists, pointsInCluster, pointIndices);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return;
    }
    hipcub::KeyValuePair<int, double> *h_argmin = new hipcub::KeyValuePair<int, double>();
    hipcub::KeyValuePair<int, double> *d_argmin;
    hipMalloc(&d_argmin, sizeof(hipcub::KeyValuePair<int, double>));
    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, dists, d_argmin, pointsInCluster);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, pointsInCluster * sizeof(double));

    // Run argmin-reduction
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, dists, d_argmin, pointsInCluster);
    setCoords<<<1,1>>>(clusterId, d_argmin, medoids, allPoints, pointIndices);
}


