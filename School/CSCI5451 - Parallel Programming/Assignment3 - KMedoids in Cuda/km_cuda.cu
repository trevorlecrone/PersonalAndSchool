#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 199309L
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <time.h>
#include <vector>
#include <limits>
#include "km_kernels.cu"


/**
* @brief Return the number of seconds since an unspecified time (e.g., Unix
*        epoch). This is accomplished with a high-resolution monotonic timer,
*        suitable for performance timing.
*
* @return The number of seconds.
*/
static inline double monotonic_seconds()
{
  /* Linux systems */
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

/**
* @brief Output the seconds elapsed while clustering.
*
* @param seconds Seconds spent on k-medoids clustering, excluding IO.
*/
static void print_time(double const seconds)
{
  printf("k-medoids clustering time: %0.04fs\n", seconds);
}

//Parses input file
double* ParseFile(char* fileName_, int &numPoints, int &dimension) {
    FILE *fp;
    // need to find length to end of each line
    char line[10000];

    const char* fileName = fileName_;

    // open the file containing the scene description
    fp = fopen(fileName, "r");

    // check for errors in opening the file
    if (fp == NULL) {
        printf("Can't open file '%s'\n", fileName);
        double* fail = new double[1];
        fail[0] = -1.0f;
        return fail;  //Exit
    }
    
    //get info from first line
    fgets(line, 10000, fp);
    sscanf(line, "%d %d", &numPoints, &dimension);
    double* vals = new double[numPoints * dimension];

    int currentPosition = 0;
    while (currentPosition < numPoints) {
        fgets(line, 10000, fp);
        char* target = strtok(line, " ");;
        for(int i = 0; i < dimension; i++) {
            sscanf(target, "%lf", &vals[(currentPosition * dimension) + i]);
            target = strtok(NULL, " ");
        }
        currentPosition++;
    }
    fclose(fp);
    return vals;
}

int main(int argc, char** argv){

	// parse arguments
    char* fileName = argv[1];
    int numGroups = atoi(argv[2]);
    int numBlocks = atoi(argv[3]);
    int threadsPerBlock = atoi(argv[4]);

    int numPoints;
    int dimension;
    bool CALC_BLOCKS = (numBlocks == -1);
    double* allCoordComponents = ParseFile(fileName, numPoints, dimension);

    std::vector<Point> allPoints;

    for(int i = 0; i < numPoints; i++) {
        double* pointCoords = new double[dimension];
        memcpy(pointCoords, &allCoordComponents[i * dimension],  dimension * sizeof(double));
        allPoints.push_back(*(new Point(dimension, pointCoords)));
    }

    Point* pointsAsArray = allPoints.data();
    allPoints.clear();

    Point* cudaPoints;
    int pointsMemSize = (numPoints * sizeof(Point));
    hipMalloc((void**) &cudaPoints, pointsMemSize);
    hipMemcpy(cudaPoints, pointsAsArray, pointsMemSize, hipMemcpyHostToDevice);
    printf("cudaMemCopy done\n");

    hipDeviceSynchronize();
    double start = monotonic_seconds();
    int* medoidInds = new int[numPoints]();
    Point* medoids = new Point[numGroups];
    for (int i = 0; i < numGroups; i++) {
        medoids[i] = allPoints[i];
    }
    delete[] allCoordComponents;

    int* cudaMedoidInds;
    Point* cudaMedoids;
    hipMalloc(&cudaMedoidInds, numPoints * sizeof(int));
    hipMemcpy(cudaMedoidInds, medoidInds, numPoints * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&cudaMedoids, numGroups * sizeof(Point));
    hipMemcpy(cudaMedoids, medoids, numGroups * sizeof(Point), hipMemcpyHostToDevice);
    
    //printf("Cuda mem copied\n");
    hipDeviceSynchronize();
    double start_iter = monotonic_seconds();
    bool converged = false;
    if(CALC_BLOCKS) {
        numBlocks = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
    }
    callClusterPointsKernel(numGroups, numPoints, cudaMedoids, cudaPoints, dimension, cudaMedoidInds, converged, numBlocks, threadsPerBlock);
    hipDeviceSynchronize();
    double stop_iter = monotonic_seconds();
    printf("initial clustering time: %0.04fs\n", stop_iter - start_iter);
    //max iterations = 20
    for (int i = 0; i < 20; i++) {
        converged = false;
        hipDeviceSynchronize();
        double start_medoid = monotonic_seconds();
        for (int j = 0; j < numGroups; j++) {
            thrust::device_vector<int> cudaPointFlags(numPoints);
            if(CALC_BLOCKS) {
                numBlocks = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
            }
            callFlagPointsInClusterKernel(numPoints, j, cudaMedoidInds, cudaPointFlags, numBlocks, threadsPerBlock);

            int pointsInCluster = thrust::reduce(cudaPointFlags.begin(), cudaPointFlags.end(), 0, thrust::plus<int>());
            thrust::device_vector<int> indices(pointsInCluster);
            // compute indices of nonzero elements
            using IndexIterator = thrust::device_vector<int>::iterator;
            IndexIterator indices_end = thrust::copy_if(thrust::make_counting_iterator(0), thrust::make_counting_iterator(numPoints), cudaPointFlags.begin(), indices.begin(), thrust::identity<int>());
            //printf("gotIndices");                
            double* cudaDists;
            hipMalloc(&cudaDists, pointsInCluster * sizeof(double));
            if(CALC_BLOCKS) {
                numBlocks = std::max(1, (pointsInCluster + threadsPerBlock - 1) / threadsPerBlock);
            }
            callGenerateDistsAndAssignMedoidsKernel(dimension, j, cudaPoints, cudaMedoids, cudaDists, pointsInCluster, indices, numBlocks, threadsPerBlock);
            hipFree(cudaDists);
        }
        hipDeviceSynchronize();
        double stop_medoid = monotonic_seconds();
        hipDeviceSynchronize();
        double start_iter = monotonic_seconds();
        if(CALC_BLOCKS) {
            numBlocks = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
        }
        callClusterPointsKernel(numGroups, numPoints, cudaMedoids, cudaPoints, dimension, cudaMedoidInds, converged, numBlocks, threadsPerBlock);
        hipDeviceSynchronize();
        double stop_iter = monotonic_seconds();
        printf("medoid calculation time: %0.04fs\n", stop_medoid - start_medoid);
        printf("iteration clustering time: %0.04fs\n", stop_iter - start_iter);
        if(converged) {
            i += 20;
        }
    }
    hipDeviceSynchronize();
    double stop = monotonic_seconds();
    double clusteringTime = stop - start;
    print_time(clusteringTime);

    hipMemcpy(medoids, cudaMedoids, numGroups * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(medoidInds, cudaMedoidInds, numPoints * sizeof(int), hipMemcpyDeviceToHost);

    std::ofstream clustersFile;
    clustersFile.open("clusters.txt");
    for (int i = 0; i < numPoints; i++) {
        clustersFile << medoidInds[i] << "\n";
    }
    clustersFile.close();

    std::ofstream medoidFile;
    medoidFile.open("medoids.txt");
    medoidFile << std::fixed << std::setprecision(8);
    for (int i = 0; i < numGroups; i++) {
        for (int j = 0; j < dimension - 1; j++) {
            double val = medoids[i].coordinates[j];
            if(val > 1.0 || val < -1.0) {
                medoidFile  << std::fixed << std::setprecision(8) << val << " ";
            }
            else {
                medoidFile.unsetf(std::ios_base::fixed);
                medoidFile << val << " ";
            }
        }
        double fVal = medoids[i].coordinates[dimension - 1];
        if(fVal > 1.0 || fVal < -1.0) {
            medoidFile  << std::fixed << std::setprecision(8) << fVal << "\n";
        }
        else {
            medoidFile.unsetf(std::ios_base::fixed);
            medoidFile << fVal << "\n";
        }
    }
    medoidFile.close();
}